#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/bgr_to_hsv_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void BgrToHsvForward(const int num, const int size, const Dtype* bottom_data, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, num * size) {
    int n = index / size;
    int i = index % size;

    Dtype r, g, b, h, s, v, delta, out0;
    b = bottom_data[n * 3 * size + 0 * size + i];
    g = bottom_data[n * 3 * size + 1 * size + i];
    r = bottom_data[n * 3 * size + 2 * size + i];

    v = max(r, max(g, b));

    delta = v - min(r, min(g, b));

    if (v == 0) {
      s = 0.0;
    } else {
      s = delta / v;
    }

    if (r == v) {
      out0 = (g - b) / delta;
    } else if (g == v) {
      out0 = 2.0 + (b - r) / delta;
    } else {
      out0 = 4.0 + (r - g) / delta;
    }

    h = fmod((out0 / 6.0) + 10.0, 1.0);
    if (delta == 0) {
      h = 0.0;
    }

    top_data[n * 3 * size + 0 * size + i] = h;
    top_data[n * 3 * size + 1 * size + i] = s;
    top_data[n * 3 * size + 2 * size + i] = v;
  }
}

template <typename Dtype>
void BgrToHsvLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  const int num = bottom[0]->num();
  const int size = bottom[0]->height() * bottom[0]->width();

  const int count = num * size;
  // NOLINT_NEXT_LINE(whitespace/operators)
  BgrToHsvForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      num, size, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void BgrToHsvLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(BgrToHsvLayer);


}  // namespace caffe

