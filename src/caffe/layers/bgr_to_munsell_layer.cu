#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/bgr_to_munsell_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void BgrToMunsellForward(const int num, const int size, const Dtype* bottom_data, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, num * size) {
    int n = index / size;
    int i = index % size;

    Dtype r, g, b, h, s, v, c, delta, out0;
    b = bottom_data[n * 3 * size + 0 * size + i];
    g = bottom_data[n * 3 * size + 1 * size + i];
    r = bottom_data[n * 3 * size + 2 * size + i];

    v = max(r, max(g, b));

    delta = v - min(r, min(g, b));

    if (v == 0) {
      s = 0.0;
    } else {
      s = delta / v;
    }

    if (r == v) {
      out0 = (g - b) / delta;
    } else if (g == v) {
      out0 = 2.0 + (b - r) / delta;
    } else {
      out0 = 4.0 + (r - g) / delta;
    }

    h = fmod((out0 / 6.0) + 10.0, 1.0);
    if (delta == 0) {
      h = 0.0;
    }

    c = s * v;

    top_data[n * 3 * size + 0 * size + i] = h;
    top_data[n * 3 * size + 1 * size + i] = c;
    top_data[n * 3 * size + 2 * size + i] = v;
  }
}

template <typename Dtype>
void BgrToMunsellLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  const int num = bottom[0]->num();
  const int size = bottom[0]->height() * bottom[0]->width();

  const int count = num * size;
  // NOLINT_NEXT_LINE(whitespace/operators)
  BgrToMunsellForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      num, size, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void BgrToMunsellLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(BgrToMunsellLayer);


}  // namespace caffe

